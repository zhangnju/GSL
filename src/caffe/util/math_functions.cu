#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<long>(const int N, const long alpha, const long* X,
    long* Y) {
	NOT_IMPLEMENTED;
}



template  <typename Dtype>
__global__ void zerout_kernel(void * mutable_gpu_data, int count, Dtype thre){
	//Dtype thre = Dtype(th);
	Dtype* data_ptr_tmp =  static_cast<Dtype*>(mutable_gpu_data);
		//  for(int i=0;i<count;i++){
		//	  if(data_ptr_tmp[i]<thre && data_ptr_tmp[i]>(-thre)){
		//		  data_ptr_tmp[i]=0;
		//	  }
		//  }
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<count){
		if(data_ptr_tmp[tid]<=thre && data_ptr_tmp[tid]>=(-thre)){
			data_ptr_tmp[tid] = 0;
		}
		tid += gridDim.x*blockDim.x;
	}
}

template  <typename Dtype>
__global__ void zerout_kernel(int count, const Dtype *x, Dtype *y, Dtype thre){
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<count){
		if(x[tid]<=thre && x[tid]>=(-thre)){
			y[tid] = 0;
		}
    else {
      y[tid] = x[tid];
    }
		tid += gridDim.x*blockDim.x;
	}
}

template  <typename Dtype>
__global__ void zerout_kernel2(int count, Dtype *x, const Dtype *thresholds, int thresholds_len, Dtype weight){
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<count){
    Dtype thre = thresholds[tid%thresholds_len]*weight;
		if(x[tid]<=thre && x[tid]>=(-thre)){
			x[tid] = 0;
		}
		tid += gridDim.x*blockDim.x;
	}
}

template <typename Dtype>
void caffe_gpu_zerout(void * mutable_gpu_data, const int count, Dtype th){
	zerout_kernel<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, (Dtype *)mutable_gpu_data, (Dtype *)mutable_gpu_data, th);
}

template <typename Dtype>
void caffe_gpu_zerout(int count, const Dtype *x, Dtype *y, Dtype thre){
	zerout_kernel<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, x, y, thre);
}

template <typename Dtype>
void caffe_gpu_zerout(int count, Dtype *x, const Dtype *thresholds, int thresholds_len, Dtype weight){
	zerout_kernel2<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, x, thresholds, thresholds_len, weight);
}

template void caffe_gpu_zerout<int>(void * mutable_gpu_data, const int count, int th);
template void caffe_gpu_zerout<unsigned int>(void * mutable_gpu_data, const int count, unsigned int th);
template void caffe_gpu_zerout<long>(void * mutable_gpu_data, const int count, long th);
template void caffe_gpu_zerout<unsigned long>(void * mutable_gpu_data, const int count, unsigned long th);
template void caffe_gpu_zerout<float>(void * mutable_gpu_data, const int count, float th);
template void caffe_gpu_zerout<double>(void * mutable_gpu_data, const int count, double th);

template void caffe_gpu_zerout<int>(int count, const int *x, int *y, int th);
template void caffe_gpu_zerout<unsigned int>(int count, const unsigned int *x, unsigned int *y, unsigned int th);
template void caffe_gpu_zerout<long>(int count, const long *x, long *y, long th);
template void caffe_gpu_zerout<float>(int count, const float *x, float *y, float th);
template void caffe_gpu_zerout<double>(int count, const double *x, double *y, double th);

template void caffe_gpu_zerout<float>(int count, float *x, const float *thresholds, int thresholds_len, float weight);
template void caffe_gpu_zerout<double>(int count, double *x, const double *thresholds, int thresholds_len, double weight);

/*template  <typename Dtype>
__global__ void if_zerout_fiber_kernel(
  int I, int J, int K, const Dtype *x, Dtype * y, int mode, Dtype thre)
{
	int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if (0 == mode) {
    int nfiber = J*K;
    while (tid < nfiber) {
      int is_zero = 1;
      for (int i = 0; i < I; ++i) {
        if (x[i*J*K + tid] > thre || x[i*J*K + tid] < -thre) {
          is_zero = 0;
          break;
        }
      }

      y[tid] = is_zero;

      tid += gridDim.x*blockDim.x;
    }
  }
  else if (1 == mode) {
    int nfiber = J*K;
    while (tid < nfiber) {
      int is_zero = 1;
      for (int i = 0; i < I; ++i) {
        if (x[i*J*K + tid] > thre || x[i*J*K + tid] < -thre) {
          is_zero = 0;
          break;
        }
      }

      y[tid] = is_zero;

      tid += gridDim.x*blockDim.x;
    }
  }
  else {
  }
}*/

template  <typename Dtype>
__global__ void shrinkage_kernel(void * mutable_gpu_data, int count, Dtype thre){
	//Dtype thre = Dtype(th);
	Dtype* data_ptr_tmp =  static_cast<Dtype*>(mutable_gpu_data);
		//  for(int i=0;i<count;i++){
		//	  if(data_ptr_tmp[i]<thre && data_ptr_tmp[i]>(-thre)){
		//		  data_ptr_tmp[i]=0;
		//	  }
		//  }
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<count){
		if(data_ptr_tmp[tid]<thre && data_ptr_tmp[tid]>(-thre)){
			data_ptr_tmp[tid] = 0;
		}
        else if (data_ptr_tmp[tid] > 0) {
            data_ptr_tmp[tid] -= thre;
        }
        else {
            data_ptr_tmp[tid] += thre;
        }
		tid += gridDim.x*blockDim.x;
	}
}

template <typename Dtype>
void caffe_gpu_shrinkage(void * mutable_gpu_data, const int count, Dtype th){
	shrinkage_kernel<<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(mutable_gpu_data,  count,  th);
}

template void caffe_gpu_shrinkage<int>(void * mutable_gpu_data, const int count, int th);
template void caffe_gpu_shrinkage<unsigned int>(void * mutable_gpu_data, const int count, unsigned int th);
template void caffe_gpu_shrinkage<long>(void * mutable_gpu_data, const int count, long th);
template void caffe_gpu_shrinkage<float>(void * mutable_gpu_data, const int count, float th);
template void caffe_gpu_shrinkage<double>(void * mutable_gpu_data, const int count, double th);


template  <typename Dtype>
__global__ void if_zerout_kernel(const int n, const Dtype * x, Dtype *y, Dtype thre){
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<n){
		y[tid] = (x[tid]<=thre && x[tid]>=(-thre)) ? 1 : 0;
		tid += gridDim.x*blockDim.x;
	}
}

template  <typename Dtype>
__global__ void if_zerout_kernel(const int n, const Dtype * x, Dtype *y, const Dtype *thresholds, int thresholds_len, Dtype weight){
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<n){
    Dtype thre = thresholds[tid%thresholds_len]*weight;
		y[tid] = (x[tid]<=thre && x[tid]>=(-thre)) ? 1 : 0;
		tid += gridDim.x*blockDim.x;
	}
}

template <typename Dtype>
void caffe_gpu_if_zerout(const int n, const Dtype * x, Dtype *y, Dtype th){
	if_zerout_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, x, y, th);
}

template <typename Dtype>
void caffe_gpu_if_zerout(const int n, const Dtype * x, Dtype *y, const Dtype *thresholds, int thresholds_len, Dtype weight) {
	if_zerout_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, x, y, thresholds, thresholds_len, weight);
}

template void caffe_gpu_if_zerout<int>(const int n, const int * x, int *y, int th);
template void caffe_gpu_if_zerout<unsigned int>(const int n, const unsigned int* x, unsigned int *y, unsigned int th);
template void caffe_gpu_if_zerout<long>(const int n, const long* x, long *y, long th);

template void caffe_gpu_if_zerout<float>(const int n, const float * x, float *y, float th);
template void caffe_gpu_if_zerout<double>(const int n, const double* x, double *y, double th);

template void caffe_gpu_if_zerout<float>(const int n, const float * x, float *y, const float *thresholds, int thresholds_len, float weight);
template void caffe_gpu_if_zerout<double>(const int n, const double* x, double *y, const double *thresholds, int thresholds_len, double weight);

template  <typename Dtype>
__global__ void if_nonzerout_kernel(const int n, const Dtype * x, Dtype *y, Dtype thre){
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	while(tid<n){
		y[tid] = (x[tid]<=thre && x[tid]>=(-thre)) ? 0 : 1;
		tid += gridDim.x*blockDim.x;
	}
}

template <typename Dtype>
void caffe_gpu_if_nonzerout(const int n, const Dtype * x, Dtype *y, Dtype th){
	if_nonzerout_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(n, x, y, th);
}

template void caffe_gpu_if_nonzerout<int>(const int n, const int * x, int *y, int th);
template void caffe_gpu_if_nonzerout<unsigned int>(const int n, const unsigned int* x, unsigned int *y, unsigned int th);
template void caffe_gpu_if_nonzerout<long>(const int n, const long* x, long*y, long th);
template void caffe_gpu_if_nonzerout<unsigned long>(const int n, const unsigned long* x, unsigned long*y, unsigned long th);

template void caffe_gpu_if_nonzerout<float>(const int n, const float * x, float *y, float th);
template void caffe_gpu_if_nonzerout<double>(const int n, const double* x, double *y, double th);

//template <>
//void caffe_gpu_zerout<int>(void * mutable_gpu_data, int count, int th){
//	zerout_kernel<<<32768,256>>>(mutable_gpu_data,  count,  th);
//}
//
//template <>
//void caffe_gpu_zerout<float>(void * mutable_gpu_data, int count, float th){
//	zerout_kernel<<<32768,256>>>(mutable_gpu_data,  count,  th);
//}
//
//template <>
//void caffe_gpu_zerout<double>(void * mutable_gpu_data, int count, double th){
//	zerout_kernel<<<32768,256>>>(mutable_gpu_data,  count,  th);
//}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<long>(const int N, const long alpha, long *X) {
   NOT_IMPLEMENTED;
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<long>(const int n, const long* x, const long* y,
    long * out) {
  NOT_IMPLEMENTED;
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<long>(const int n, const long* x, long* y) {
  NOT_IMPLEMENTED;
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <typename Dtype>
__global__ void div_checkzero_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = b[index] ? (a[index] / b[index]) : Dtype(0);
  }
}

template <typename Dtype>
__global__ void inv_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = 1 / a[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div_checkzero<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_checkzero_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div_checkzero<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
	div_checkzero_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_inv<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  inv_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_inv<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  inv_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
